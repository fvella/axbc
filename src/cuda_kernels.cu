#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <mpi.h>

#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/sort.h>
#include "cub/device/device_scan.cuh"
#include "cudamacro.h"
#include "bc2d.h"
using namespace hipcub;

// BEST  DRAKE 128 1 2 
// BEST PizDaint 2 2  256
#define THREADS (128)
#define ROWXTH 2
#define ROWXTHD 1
#define THRUST 1
__device__ __constant__ LOCINT dN;
__device__ __constant__ LOCINT drow_bl;
__device__ __constant__ LOCINT dcol_bl;
__device__ __constant__ LOCINT drow_pp;

__device__ __constant__ int dC;
__device__ __constant__ int dR;
__device__ __constant__ int dmyrow;
__device__ __constant__ int dmycol; 

__device__ LOCINT dnfrt;

__device__ LOCINT d_reach_v0; 



static LOCINT	*d_msk=NULL;
static int	*d_lvl=NULL;

static LOCINT	*d_col=NULL;
static LOCINT	*d_row=NULL;
static LOCINT	*d_deg=NULL;

static LOCINT	*d_rbuf=NULL;
static LOCINT	*d_cbuf=NULL;
static LOCINT	*d_cbuf_start=NULL;

static LOCINT	*d_sbuf=NULL;
static uint32_t	*d_snum=NULL;

static LOCINT   *d_frt=NULL;
static LOCINT   *d_frt_start=NULL;
static LOCINT   *d_frt_sig=NULL;


static LOCINT   *d_sig=NULL;

static LOCINT   *d_tmp_sig=NULL;
static LOCINT	*d_rbuf_sig=NULL;
static LOCINT	*d_sbuf_sig=NULL;

static float    *d_delta=NULL;

static float    *d_fsbuf=NULL;
static float    *d_frbuf=NULL;
static float 	*d_bc=NULL;
static LOCINT   *d_reach= NULL;
static LOCINT   *d_all = NULL;


#ifndef THRUST
static size_t cubtmp_sz=0;
static LOCINT *d_cubtmp= NULL;
#endif
hipEvent_t     start, stop;
hipStream_t    stream[2];






FILE *Fopen(const char *path, const char *mode) {

	FILE *fp = NULL;
	fp = fopen(path, mode);
	if (!fp) {
		fprintf(stderr, "Cannot open file %s...\n", path);
		exit(EXIT_FAILURE);
	}
	return fp;
}

void dump_device_array(const char *name, LOCINT *d_arr, int n) {

	FILE	 *fp=NULL;
	char	 fname[MAX_LINE];
	int i;
	LOCINT *in;

	snprintf(fname, MAX_LINE, "%s_%d", name, myid);
	fp = Fopen(fname, "a");

	in = (LOCINT *)Malloc(n*sizeof(*in));
	MY_CUDA_CHECK( hipMemcpy(in, d_arr, n*sizeof(*in), hipMemcpyDeviceToHost) );

	for (i = 0; i < n ; i++)
	   fprintf(fp, " %d,", in[i]);

	fprintf(fp, "\n");
	fclose(fp);
	free(in);
	return;

}

void dump_array2(int *arr, int n, const char *name) {

	if (outdebug==NULL) return;
	int i;
	fprintf(outdebug, "%s - %d\n",name, n);

	for (i = 0; i < n ; i++)
	   fprintf(outdebug, " %d,", arr[i]);

	fprintf(outdebug, "\n");
	return;

}

void dump_uarray2(LOCINT *arr, int n, const char *name) {

	if (outdebug==NULL) return;
	int i;
	fprintf(outdebug, "%s - %d\n",name, n);

	for (i = 0; i < n ; i++)
	   fprintf(outdebug, " %d,", arr[i]);

	fprintf(outdebug, "\n");
	return;
}

void dump_farray2(float *arr, int n, const char *name) {

	if (outdebug==NULL) return;
	int i;
	fprintf(outdebug, "%s - %d\n",name, n);
	for (i = 0; i < n ; i++)
	   fprintf(outdebug, " %f,", arr[i]);

	fprintf(outdebug, "\n");
	return;

}

void dump_device_array2(int *d_arr, int n, const char * name) {

	if (outdebug==NULL) return;
	int i;
	int *in;

	fprintf(outdebug, "%s - %d\n",name, n);

	in = (int *)Malloc(n*sizeof(*in));
	MY_CUDA_CHECK( hipMemcpy(in, d_arr, n*sizeof(*in), hipMemcpyDeviceToHost) );

	for (i = 0; i < n ; i++)
	   fprintf(outdebug, " %d,", in[i]);

	fprintf(outdebug, "\n");
	fflush(outdebug);
	free(in);
	return;

}

void dump_device_uarray2(LOCINT *d_arr, int n, const char * name) {

	if (outdebug==NULL) return;
	int i;
	LOCINT *in;

	fprintf(outdebug, "%s - %d\n",name, n);

	in = (LOCINT *)Malloc(n*sizeof(*in));
	MY_CUDA_CHECK( hipMemcpy(in, d_arr, n*sizeof(*in), hipMemcpyDeviceToHost) );

	for (i = 0; i < n ; i++)
	   fprintf(outdebug, " %d,", in[i]);

	fprintf(outdebug, "\n");
	fflush(outdebug);
	free(in);
	return;

}

void dump_device_farray2(float *d_arr, int n, const char * name) {

	if (outdebug==NULL) return;
	int i;
	float *in;

	fprintf(outdebug, "%s - %d\n",name, n);

	in = (float *)Malloc(n*sizeof(*in));
	MY_CUDA_CHECK( hipMemcpy(in, d_arr, n*sizeof(*d_arr), hipMemcpyDeviceToHost) );

	for (i = 0; i < n ; i++)
	   fprintf(outdebug, " %f,", in[i]);

	fprintf(outdebug, "\n");
	fflush(outdebug);
	free(in);
	return;

}


// returns the index of the maximum i | v[i] <= val
__device__ LOCINT bmaxlt(const LOCINT *__restrict__ v, LOCINT num, LOCINT val) {

	LOCINT	min = 0;
	LOCINT	max = num-1;
	LOCINT	mid = max >> 1;

	while(min <= max) {

		if (v[mid] == val)	return mid;
		if (v[mid]  < val)	min = mid+1;
		else			max = mid-1;
		mid = (max>>1)+(min>>1)+((min&max)&1); //(max + min) >> 1
	}
	return mid;
} 
	
__global__ void read_edge_count(const LOCINT *__restrict__ deg, const LOCINT *__restrict__ rbuf, LOCINT n, LOCINT *cbuf) {
	
	const uint32_t tid = blockDim.x*blockIdx.x + threadIdx.x;

	if (tid >= n) return;
	cbuf[tid] = deg[rbuf[tid]];
	return;
}

/*
 * write_sigma (d_sbuf+i*ld, d_sig, d_tmp_sig, snum[i], d_sbuf_sig+i*ld);
 */
__global__ void write_sigma(const LOCINT *__restrict__ sbuf, const LOCINT *__restrict__ sigma,
		                    LOCINT * tmp_sig, LOCINT n, LOCINT *sbuf_sig) {
	
	const uint32_t tid = blockDim.x*blockIdx.x + threadIdx.x;

	if (tid >= n) return;

	sbuf_sig[tid] = sigma[sbuf[tid]] + tmp_sig[sbuf[tid]]; // Calculate the total sigma and prepare for sending
	tmp_sig[sbuf[tid]] = 0; // So we already have the array zero for next round

	return;
}



__global__ void update_bc(const float *__restrict__ delta, int r0, float p, LOCINT n,  float *bc, LOCINT *reach, const uint64_t nvisited) {

	const uint32_t tid = blockDim.x*blockIdx.x + threadIdx.x;

	if (tid >= n) return;
	
	if (r0 == tid)  {
		if (d_reach_v0 > 0) bc[tid] = __ldg(&bc[tid])+ (d_reach_v0)*(nvisited-2);
		return;
	}

    // in verita' non e' un problema ma la cosa e' ingannevole:
    // trace dei caller row_pp -> ncol -> n
	//bc[tid] += delta[tid]*(reach[r0]+1); ///2.0f; NON FUNZIONA SE LO FACCIAMO QUI
    	bc[tid] = __ldg(&bc[tid]) + ( delta[tid]*(d_reach_v0 + 1)/p);
//	if (tid == 0) printf("%f\n", p);
	return;
}


void update_bc_cuda(LOCINT v0, float p,int ncol, const uint64_t __restrict__ nvisited) {
	// v0 is the GLOBAL root vertex
    //printf("%s at %d : %d  v: %d   proc: %d\n", __func__, __LINE__, ncol, GI2LOCI(v0),myid);
	int r0 = -1;
	if (GI2PI(v0) == myrow) {
		r0 = GI2LOCI(v0);
	}
	// dump_array2((int*)&v0, 1, "ROOT_VERTEX");
	// dump_array2(&r0, 1, "LOCAL_ROW_INDEX");
    //dump_device_farray2(d_delta, ncol, "FINAL_DELTA");
    // dump_device_farray2(d_bc, ncol, "UPDATE_BC");
	update_bc<<<(ncol+THREADS-1)/THREADS, THREADS>>>(d_delta, r0, p,ncol, d_bc, d_reach, nvisited);
    //dump_device_farray2(d_bc, ncol, "FINAL_BC");
}
void sort_by_degree(LOCINT *deg, LOCINT *bc_order){
//	uint64_t i = 0;
/*	for (i =0 ; i < N ; i++){
			printf("node-id %d degree %d\n", bc_order[i], deg[GJ2LOCJ(i)]);
		bc_order [i] = i; //naive iniit 

	}
*/
	thrust::sort_by_key(deg, deg + N, bc_order);
	//after sort 
/*	printf("\n\nAFTER SORT\n\n");
	for (i =0 ; i < N ; i++){
			printf("node-id %d degree %d\n", bc_order[i], deg[GJ2LOCJ(i)]);

	}
*/
}

__inline__ __device__	int warpReduceSum(int val) {
	for (int offset = warpSize/2; offset > 0; offset /= 2) 
    		val += __shfl_down(val, offset);
	return val;
}

__inline__ __device__ int blockReduceSum(int val) {

 	static __shared__ int shared[32]; 	
	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;
  	val = warpReduceSum(val); 
  	if (lane==0) shared[wid]=val;	  __syncthreads();              
  	val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;
  	if (wid==0) val = warpReduceSum(val); //Final reduce within first warp
  	return val;
}

__global__ void deviceReduceKernel(const LOCINT *__restrict__ in, LOCINT* out, int N, const int * __restrict__ cond) {
	LOCINT sum = 0;
//	int p= 0;
	const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = tid; i < N; i += THREADS/4 * THREADS/4) {
		//if (cond[i] > 0) sum+= in[i];
    		sum += in[i]*(cond[i] > 0 );
//		p = in[i];
//		if ( cond[i] == -1 ) p=0;
//               sum += p;


	}
	sum = blockReduceSum(sum);
	if (threadIdx.x == 0)atomicAdd(out, sum);
}



void pre_update_bc_cuda(LOCINT *reach, LOCINT v0, LOCINT *all){


// all is the number of removed 1-degree in the same connected component 
/*	LOCINT test = 0;	
	int j = 0;
        for(j = 0; j < row_pp; j++){
		if (lvl[j]>0) test += reach[j]; 
	}
*/
	hipMemsetAsync(d_all,0,sizeof(LOCINT));
  	deviceReduceKernel<<<THREADS/4, THREADS/4>>>(d_reach, d_all, row_pp, d_lvl);
	hipMemcpy(all,d_all,sizeof(int),hipMemcpyDeviceToHost);
//	if (test != all[0])
//	printf("reduction: cpu = %d |  gpu = %d  \n",test,all[0]);
}

/*
 * write_delta<<<(ncol+THREADS-1)/THREADS, THREADS>>>(d_rbuf, d_sig, d_reach, d_frbuf, ncol, d_fsbuf);
 */
__global__ void write_delta(const LOCINT *__restrict__ frt, const LOCINT *__restrict__ sigma,
		                    const LOCINT *__restrict__ reach,
						   float  *rbuf, LOCINT n,  float *sbuf) {

	const uint32_t tid = blockDim.x*blockIdx.x + threadIdx.x;
	LOCINT i;
	if (tid >= n) return;

	if (CUDA_ISMYCOLL(frt[tid])) {
		// No race condition since a node appears only ones in the frontier
		// Calculate delta only for my own vertices
		// Here delta is updated using row index
		i = CUDA_MYLOCJ2LOCI(frt[tid]);
		//sbuf[i] = rbuf[tid] * sigma[i] + reach[i]; // add reach[i]
                sbuf[i] = rbuf[tid] * sigma[i];

	    // Copy back the value into the send-receive buffer
	    //srbuf[tid] = delta[i] ;
	}
	rbuf[tid] = 0;

}

LOCINT write_delta_cuda(LOCINT ncol, float *hRFbuf, float *hSFbuf) {

	float	et=0;
	TIMER_DEF(1);

	TIMER_START(1);

	// Reset send buffer

	MY_CUDA_CHECK( hipMemset(d_fsbuf, 0, row_pp*sizeof(*d_fsbuf)) );
	if (!ncol) {
		TIMER_STOP(1);
		goto out;
	}
	// Copy receive buffer into device memory
	MY_CUDA_CHECK( hipMemcpy(d_frbuf, hRFbuf , ncol*sizeof(*hRFbuf), hipMemcpyHostToDevice ));

	TIMER_STOP(1);

	MY_CUDA_CHECK( hipEventRecord(start, 0) );

	// READ_DFRT
	write_delta<<<(ncol+THREADS-1)/THREADS, THREADS>>>(d_rbuf, d_sig, d_reach, d_frbuf, ncol, d_fsbuf);

	// Here we have d_delta updated
	MY_CUDA_CHECK( hipEventRecord(stop, 0) );
	MY_CHECK_ERROR("write_delta");
	MY_CUDA_CHECK( hipEventSynchronize(stop) );
	MY_CUDA_CHECK( hipEventElapsedTime(&et, start, stop) );
	//dump_farray2(hSRbuf, row_pp, "write_delta_cuda::hSRbuf");
	//dump_device_farray2(d_delta, row_pp, "write_delta_cuda::d_delta");
	//dump_device_farray2(d_fsbuf, row_pp, "write_delta_cuda::d_fsbuf");
	//Copy delta into send-receive buffer
out:
	MY_CUDA_CHECK( hipMemcpy(hSFbuf, d_fsbuf, MAX(row_pp,col_bl)*sizeof(*hSFbuf), hipMemcpyDeviceToHost ));
    //dump_farray2(hSFbuf, row_pp, "write_delta_cuda::DELTA");

   return ncol;
}

__global__ void scan_col(const LOCINT *__restrict__ row,  const LOCINT *__restrict__ col, LOCINT nrow,
			                const LOCINT *__restrict__ rbuf, const LOCINT *__restrict__ rbuf_sig,
			                const LOCINT *__restrict__ cbuf, LOCINT ncol,
			             LOCINT *msk, int *lvl, LOCINT* tmp_sig, int level,
			             LOCINT *sbuf, uint32_t *snum) {
	
	// This processes ROWXTH elements together
	LOCINT r[ROWXTH];
	LOCINT c[ROWXTH]; // Vertex in the current frontier
	LOCINT s[ROWXTH]; // Sigma of the vertex in the current frontier
	LOCINT m[ROWXTH], q[ROWXTH], i[ROWXTH];

	const uint32_t tid = (blockDim.x*blockIdx.x + threadIdx.x)*ROWXTH;

	if (tid >= nrow) return;

	// Use binary search to calculate predecessor position in the rbuf array
	i[0] = bmaxlt(cbuf, /*(tid<ncol)?tid+1:*/ncol, tid);

	for(; (i[0]+1 < ncol) && (tid+0) >= cbuf[i[0]+1]; i[0]++); // Here increment i[0]
	#pragma unroll
	for(int k = 1; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		for(i[k]=i[k-1]; (i[k]+1 < ncol) && (tid+k) >= cbuf[i[k]+1]; i[k]++); // Here increment i[k]
	}

	#pragma unroll
	for(int k = 0; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		c[k] = __ldg(&rbuf[i[k]]); 
		s[k] = __ldg(&rbuf_sig[i[k]]);
	} //c[k] is the predecessor, s[k] is its sigma

	// Here r[k] corresponds to the row and from it I can determine the processor hproc
	// col[c[k]] offset in the CSC where neightbour of c[k] starts
	// row[col[c[k]] first neightbour
	// r[k] this is the visited vertex
	#pragma unroll
	for(int k = 0; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		r[k] = row[col[c[k]]+(tid+k)-cbuf[i[k]]];  // new vertex
	}

	#pragma unroll
	for(int k = 0; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		m[k] = ((LOCINT)1) << (r[k]%BITS(msk));  // its mask value
	}
	
	#pragma unroll
	for(int k = 0; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		if (__ldg(&msk[r[k]/BITS(msk)])&m[k]) //continue;
			q[k] = m[k]; // the if below will eval to false...
		else
			q[k] = atomicOr(msk+r[k]/BITS(msk), m[k]);
		
		int hproc = r[k] / drow_bl; // Calculate the Processor
		if (!(m[k]&q[k])) {  // New vertex
			uint32_t off = atomicInc(snum+hproc, 0xFFFFFFFF); // Offset // Increment + 1
			// This is the GLOBAL VERTEX !!
			sbuf[hproc*drow_bl + off] = r[k]; // Copy the new discovered vertex into the sbuf for sending
			if (hproc == dmycol){
				 lvl[r[k]] = level; // Update level
			}
		}

		// Update sigma temp regardless it was already visited or not
		atomicAdd(tmp_sig + r[k], s[k]);

	} // end for over k
	return;
}

__global__ void scan_col_mono(const LOCINT *__restrict__ row,  const LOCINT *__restrict__ col, LOCINT nrow, 
                                const LOCINT *__restrict__ rbuf, 
                                const LOCINT *__restrict__ cbuf, LOCINT ncol, 
                                LOCINT *msk, int *lvl, LOCINT* sig, int level, 
                                LOCINT *sbuf, uint32_t *snum) {

	// This processes ROWXTH elements together
	LOCINT r[ROWXTH];
	LOCINT c[ROWXTH]; // Vertex in the current frontier
	LOCINT s[ROWXTH]; // Sigma of the vertex in the current frontier
	LOCINT m[ROWXTH], q[ROWXTH], i[ROWXTH];

	const uint32_t tid = (blockDim.x*blockIdx.x + threadIdx.x)*ROWXTH;

	if (tid >= nrow) return;

	// Use binary search to calculate predecessor position in the rbuf array
	i[0] = bmaxlt(cbuf, /*(tid<ncol)?tid+1:*/ncol, tid);

	for(; (i[0]+1 < ncol) && (tid+0) >= cbuf[i[0]+1]; i[0]++); // Here increment i[0]
	#pragma unroll
	for(int k = 1; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		for(i[k]=i[k-1]; (i[k]+1 < ncol) && (tid+k) >= cbuf[i[k]+1]; i[k]++); // Here increment i[k]
	}

	#pragma unroll
	for(int k = 0; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		c[k] = __ldg(&rbuf[i[k]]); 
		s[k] = __ldg(&sig[c[k]]); 
	} //c[k] is the predecessor, s[k] is its sigma

	// Here r[k] corresponds to the row and from it I can determine the processor hproc
	// col[c[k]] offset in the CSC where neightbour of c[k] starts
	// row[col[c[k]] first neightbour
	// r[k] this is the visited vertex
	#pragma unroll
	for(int k = 0; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		r[k] = row[col[c[k]]+(tid+k)-cbuf[i[k]]];  // new vertex
	}

	#pragma unroll
	for(int k = 0; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		m[k] = ((LOCINT)1) << (r[k]%BITS(msk));  // its mask value
	}

	#pragma unroll
	for(int k = 0; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		if (__ldg(&msk[r[k]/BITS(msk)])&m[k]) //continue;
			q[k] = m[k]; // the if below will eval to false...
		else
			q[k] = atomicOr(msk+r[k]/BITS(msk), m[k]);

		if (!(m[k]&q[k])) {  // New vertex
			uint32_t off = atomicInc(snum, 0xFFFFFFFF); // Offset // Increment + 1
			// This is the GLOBAL VERTEX !!
			sbuf[off] = r[k]; // Copy the new discovered vertex into the sbuf for sending
  		    	lvl[r[k]] = level; // Update level
		}

		if (__ldg(&lvl[r[k]]) == -1 || __ldg(&lvl[r[k]]) == level) { // Order in the OR is important!			// Update sigma
            // Update sigma
			atomicAdd(sig + r[k], s[k]);
                }

	} // end for over k
	return;
}

__global__ void scan_col_mono2(const LOCINT *__restrict__ row,  const LOCINT *__restrict__ col, LOCINT nrow, 
                                const LOCINT *__restrict__ rbuf, 
                                const LOCINT *__restrict__ cbuf, LOCINT ncol, 
                                LOCINT *msk, int *lvl, LOCINT* sig, int level, 
                                LOCINT *sbuf, uint32_t *snum) {

	// This processes ROWXTH elements together
	LOCINT r[ROWXTH];
	LOCINT c[ROWXTH]; // Vertex in the current frontier
	LOCINT m[ROWXTH], q[ROWXTH], i[ROWXTH];

	const uint32_t tid = (blockDim.x*blockIdx.x + threadIdx.x)*ROWXTH;

	if (tid >= nrow) return;

	// Use binary search to calculate predecessor position in the rbuf array
	i[0] = bmaxlt(cbuf, /*(tid<ncol)?tid+1:*/ncol, tid);

	for(; (i[0]+1 < ncol) && (tid+0) >= cbuf[i[0]+1]; i[0]++); // Here increment i[0]
	#pragma unroll
	for(int k = 1; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		for(i[k]=i[k-1]; (i[k]+1 < ncol) && (tid+k) >= cbuf[i[k]+1]; i[k]++); // Here increment i[k]
	}

	#pragma unroll
	for(int k = 0; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		c[k] = __ldg(&rbuf[i[k]]); 
	} //c[k] is the predecessor, s[k] is its sigma

	// Here r[k] corresponds to the row and from it I can determine the processor hproc
	// col[c[k]] offset in the CSC where neightbour of c[k] starts
	// row[col[c[k]] first neightbour
	// r[k] this is the visited vertex
	#pragma unroll
	for(int k = 0; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		r[k] = row[col[c[k]]+(tid+k)-cbuf[i[k]]];  // new vertex
	}

	#pragma unroll
	for(int k = 0; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		m[k] = ((LOCINT)1) << (r[k]%BITS(msk));  // its mask value
	}

	#pragma unroll
	for(int k = 0; k < ROWXTH; k++) {
		if (tid+k >= nrow) break;
		if (__ldg(&msk[r[k]/BITS(msk)])&m[k]) //continue;
			q[k] = m[k]; // the if below will eval to false...
		else
			q[k] = atomicOr(msk+r[k]/BITS(msk), m[k]);

		if (!(m[k]&q[k])) {  // New vertex
			uint32_t off = atomicInc(snum, 0xFFFFFFFF); // Offset // Increment + 1
			// This is the GLOBAL VERTEX !!
			sbuf[off] = r[k]; // Copy the new discovered vertex into the sbuf for sending
  		    	lvl[r[k]] = level; // Update level
		}

		const int l = __ldg(&lvl[r[k]]);
		if (l == -1 || l == level) { // Order in the OR is important!			// Update sigma
            // Update sigma
			atomicAdd(sig + r[k], __ldg(&sig[c[k]]));
                }

	} // end for over k
	return;
}


__global__ void scan_frt(const LOCINT *__restrict__ row,   const LOCINT *__restrict__ col, LOCINT nrow,
			             const LOCINT *__restrict__ rbuf,  const LOCINT *__restrict__ cbuf, LOCINT ncol,
			             const LOCINT *__restrict__ sigma, const float  *__restrict__ delta,
			             const int *__restrict__ lvl, int depth, float *srbuf) {

	// This processes ROWXTH elements together
	LOCINT r[ROWXTHD];
	LOCINT c[ROWXTHD]; // Vertex in the current frontier
	LOCINT i[ROWXTHD];
	float a;

	const uint32_t tid = (blockDim.x*blockIdx.x + threadIdx.x)*ROWXTHD;

	if (tid >= nrow) return;

	// Use binary search to calculate predecessor position in the rbuf array
	i[0] = bmaxlt(cbuf, /*(tid<ncol)?tid+1:*/ncol, tid);

	for(; (i[0]+1 < ncol) && (tid+0) >= cbuf[i[0]+1]; i[0]++); // Here increment i[0]
	#pragma unroll
	for(int k = 1; k < ROWXTHD; k++) {
		if (tid+k >= nrow) break;
		for(i[k]=i[k-1]; (i[k]+1 < ncol) && (tid+k) >= cbuf[i[k]+1]; i[k]++); // Here increment i[k]
	}

	#pragma unroll
	for(int k = 0; k < ROWXTHD; k++) {
		if (tid+k >= nrow) break;
		c[k] = rbuf[i[k]];  } //c[k] is the vertex in the input buffer

	// Here r[k] corresponds to the row and from it I can determine the processor hproc
	// col[c[k]] offset in the CSC where neightbour of c[k] starts
	// row[col[c[k]] first neightbour
	// r[k] this is the visited vertex
	#pragma unroll
	for(int k = 0; k < ROWXTHD; k++) {
		if (tid+k >= nrow) break;
		r[k] = row[col[c[k]]+(tid+k)-cbuf[i[k]]];  // new vertex
	}

	#pragma unroll
	for (int k = 0; k < ROWXTHD; k++) {
		if (tid+k >= nrow) break;

		if (lvl[r[k]] == depth+1) { // this is a successor
			// sigma and delta are indexed by row
			a = (delta[r[k]] + 1)/sigma[r[k]];
			// IN SINGLE DEVICE we could multiply a * sigma[c[k]]
			 // Need to add into the SRbuffer using the same index used to access rbuf
			atomicAdd(srbuf+i[k], a);
		}
	} // end for over k
	return;
}




__global__ void scan_frt_mono(const LOCINT *__restrict__ row,   const LOCINT *__restrict__ col, LOCINT nrow,
			                  const LOCINT *__restrict__ rbuf,  const LOCINT *__restrict__ cbuf, LOCINT ncol,
			                  const LOCINT *__restrict__ sigma, float  *delta,
			                  const int *__restrict__ lvl, int depth) {

	// This processes ROWXTH elements together
	LOCINT r[ROWXTHD];
	LOCINT c[ROWXTHD]; // Vertex in the current frontier
	LOCINT i[ROWXTHD];
	float a;

	const uint32_t tid = (blockDim.x*blockIdx.x + threadIdx.x)*ROWXTHD;

	if (tid >= nrow) return;

	// Use binary search to calculate predecessor position in the rbuf array
	i[0] = bmaxlt(cbuf, ncol, tid);

	for(; (i[0]+1 < ncol) && (tid+0) >= cbuf[i[0]+1]; i[0]++); // Here increment i[0]
	#pragma unroll
	for(int k = 1; k < ROWXTHD; k++) {
		if (tid+k >= nrow) break;
		for(i[k]=i[k-1]; (i[k]+1 < ncol) && (tid+k) >= __ldg(&cbuf[i[k]+1]); i[k]++); // Here increment i[k]
	}

	#pragma unroll
	for(int k = 0; k < ROWXTHD; k++) {
		if (tid+k >= nrow) break;
		c[k] = __ldg(&rbuf[i[k]]);  } //c[k] is the vertex in the input buffer

	// Here r[k] corresponds to the row and from it I can determine the processor hproc
	// col[c[k]] offset in the CSC where neightbour of c[k] starts
	// row[col[c[k]] first neightbour
	// r[k] this is the visited vertex
	#pragma unroll
	for(int k = 0; k < ROWXTHD; k++) {
		if (tid+k >= nrow) break;
		r[k] = row[col[c[k]]+(tid+k)-cbuf[i[k]]];  // new vertex
	}

	#pragma unroll
	for (int k = 0; k < ROWXTHD; k++) {
		if (tid+k >= nrow) break;

		if (lvl[r[k]] == depth+1) { // this is a successor
			// sigma and delta are indexed by row
			a = (__ldg(&delta[r[k]]) + 1)/sigma[r[k]]*sigma[c[k]];
			// IN SINGLE DEVICE we multiply a * sigma[c[k]]
		    	// Need to add into the SRbuffer using the same index used to access rbuf
			atomicAdd(delta+c[k], a);
		}
	} // end for over k
	return;
}

__global__ void append_row(const LOCINT *__restrict__ row,  const LOCINT *__restrict__ row_sig, LOCINT n,
			               const LOCINT *__restrict__ cbuf, LOCINT np,
			               LOCINT *msk,  const LOCINT * __restrict__ reach, int *lvl,
			               int level, LOCINT *frt, LOCINT *tmp_sig, LOCINT * frt_sig, uint32_t *all) {

	LOCINT	 r, m, q, s;
	const uint32_t tid = blockDim.x*blockIdx.x + threadIdx.x;

	if (tid >= n) return;

	r = row[tid];
	s = row_sig[tid];

	m = ((LOCINT)1) << (r%BITS(msk));

	if (!(msk[r/BITS(msk)]&m)) {  // Check if the vertex was already visited
		q = atomicOr(msk+r/BITS(msk), m);  // Mark visited
		if (!(m&q)) { // Check if the vertex was already visited
			uint32_t off = atomicInc(&dnfrt, 0xFFFFFFFF);
			frt[off] = r;  // Still Global
			frt_sig[off] = 0;
			lvl[r] = level;
		}
	}

	if (lvl[r] == level || lvl[r] == -1) {
		// Update sigma with the value provided
		atomicAdd(tmp_sig+r, s);
	}

	return;
}

// append_sigma<<<(nfrt+THREADS-1)/THREADS, THREADS>>>(d_frt, d_sig, d_frt_sig, d_tmp_sig, nfrt);
__global__ void append_sigma(LOCINT * sbuf, LOCINT * sigma, LOCINT *sbuf_sig, LOCINT * tmp_sig, LOCINT n) {

	const uint32_t tid = blockDim.x*blockIdx.x + threadIdx.x;

	if (tid >= n) return;

	sbuf_sig[tid] = sbuf_sig[tid] + tmp_sig[sbuf[tid]];  // this is d_sbuf_sig

	sigma[sbuf[tid]] = sbuf_sig[tid];

	tmp_sig[sbuf[tid]] = 0;
	sbuf[tid] = CUDA_MYLOCI2LOCJ(sbuf[tid]); // Row index to Column Index

	return;
}

static size_t tot_dev_mem = 0;

static void *CudaMallocSet(size_t size, int val) {

        void *ptr;

        MY_CUDA_CHECK( hipMalloc(&ptr, size) );
        MY_CUDA_CHECK( hipMemset(ptr, val, size) );
        tot_dev_mem += size;

        return ptr;
}

void *CudaMallocHostSet(size_t size, int val) {

        void *ptr;

        MY_CUDA_CHECK( hipHostMalloc(&ptr, size) );
        memset(ptr, val, size);
        return ptr;
}

void CudaFreeHost(void *ptr) {

        MY_CUDA_CHECK( hipHostFree(ptr) );
        return;
}

__global__ void set_degree(LOCINT *col, LOCINT *deg, LOCINT n) {
	
	const uint32_t tid = blockDim.x*blockIdx.x + threadIdx.x;

	if (tid >= n) return;
	deg[tid] = col[tid+1] - col[tid];
	return;

}

void set_mlp_cuda(LOCINT row, int level, int sigma) {

	LOCINT v;
	MY_CUDA_CHECK( hipMemcpy(&v, d_msk+row/BITS(d_msk), sizeof(v), hipMemcpyDeviceToHost) );

	v |= (1ULL<<(row%BITS(d_msk))); 
	MY_CUDA_CHECK( hipMemcpy(d_msk+row/BITS(d_msk), &v, sizeof(*d_msk), hipMemcpyHostToDevice) );

	MY_CUDA_CHECK( hipMemcpy(d_lvl+row, &level, sizeof(level), hipMemcpyHostToDevice) );
	MY_CUDA_CHECK( hipMemcpy(d_sig+row, &sigma, sizeof(sigma), hipMemcpyHostToDevice) );

	MY_CUDA_CHECK( hipMemcpy(d_frt, &row, sizeof(row), hipMemcpyHostToDevice) );

	return;
}

__global__ void compact(LOCINT *col, LOCINT *row, LOCINT *deg, LOCINT *msk) {

	int n;
	LOCINT *v;
        int bid = threadIdx.x;
        int lid = threadIdx.x % warpSize;
        int wid = threadIdx.x / warpSize;

        int goff = 0;
        int bcount = 0;

        __shared__ uint32_t sh_v[32];

        v = row + col[blockIdx.x];
        n = deg[blockIdx.x];

        // sync()s in the loop don't cause stall because
        // every warp that cycle has either all threads
        // cycling or some cycling and some returned.
        for(; bid < n; bid += blockDim.x) {

                LOCINT	 s = v[bid];
                uint32_t m;
                int	 t;

		m = ((LOCINT)1) << (s%BITS(msk));
	        t = (msk[s/BITS(msk)]&m) == 0;

                m = __ballot(t);

                if (lid == wid) sh_v[wid] = __popc(m);
                bcount = __syncthreads_count(t);

                if (wid == 0) {
                        uint32_t k;
                        uint32_t l = sh_v[lid];
                        for (k=1; k <= 16; k <<= 1) {
                                // source th is always present so shfl
                                // never returns the calling th's var
                                int r = __shfl_up((int)l, k, 32);
                                if (lid >= k) l += r;
                        }
                        sh_v[lid] = l;
                }
                uint32_t i = __popc(m & ((1<<lid)-1));
                __syncthreads();

                uint32_t off = (!wid) ? 0 : sh_v[wid-1];
                if (t) v[goff + off + i] = s;

                goff += bcount;
        }
        if (threadIdx.x == 0) deg[blockIdx.x] = goff;
        return;
}

/*
 * scan_frt_csc_cuda(frt, ncol, depth, hRFbuf);
 */
LOCINT scan_frt_csc_cuda(const LOCINT *__restrict__ frt, int ncol, int depth, float *hSRbuf) {

#ifdef THRUST

	static	thrust::device_ptr<LOCINT> d_val(d_cbuf);
#endif
	LOCINT	i;
	int blocks, nrow=0;
	float	et=0;
	TIMER_DEF(1);
	TIMER_DEF(2);

	TIMER_START(1);
	MY_CUDA_CHECK( hipMemset(d_fsbuf, 0, ncol*sizeof(*d_fsbuf)) );

	if (!ncol) {
		TIMER_STOP(1);
		goto out;
	}

	// WRITE_DFRT
	MY_CUDA_CHECK( hipMemcpy(d_rbuf, frt, ncol*sizeof(*d_rbuf), hipMemcpyHostToDevice) );

	// calculate degree for each vertex in frt
	read_edge_count<<<(ncol+THREADS-1)/THREADS, THREADS>>>(d_deg, d_rbuf, ncol, d_cbuf);
	
	//dump_array2((uint32_t*)&ncol,1,"scan_frt_csc ncol");
	//dump_device_uarray2(d_cbuf, MAX(col_bl, C), "scan_frt_csc d_cbuf 1");

	MY_CUDA_CHECK( hipMemcpy(&i, d_cbuf+ncol-1, sizeof(*d_cbuf), hipMemcpyDeviceToHost) );
	nrow = i;

	// Prefix sum to count how many threads to launch
#ifdef THRUST
	thrust::exclusive_scan(d_val, d_val+ncol, d_val);
#else
	hipcub::DeviceScan::ExclusiveSum(d_cubtmp, cubtmp_sz, d_cbuf, d_cbuf, ncol);
#endif
	MY_CUDA_CHECK( hipMemcpy(&i, d_cbuf+ncol-1, sizeof(*d_cbuf), hipMemcpyDeviceToHost) );

	nrow += i;

	if (!nrow) {
		TIMER_STOP(1);
		goto out;
	}
	TIMER_STOP(1);

	MY_CUDA_CHECK( hipEventRecord(start, 0) );

	blocks = (((nrow+ROWXTHD-1)/ROWXTHD)+THREADS-1)/THREADS;
	//dump_device_farray2(d_delta, row_pp, "d_delta");

	scan_frt<<<blocks, THREADS>>>(d_row, d_col, nrow, d_rbuf, d_cbuf, ncol,
			                      d_sig, d_delta, d_lvl, depth, d_fsbuf);

	// Here we have d_sbuf updated with the new discovered vertices and d_tmp_sig with the local value of the accumulated sigma
	MY_CUDA_CHECK( hipEventRecord(stop, 0) );
	MY_CHECK_ERROR("scan_frt");
	MY_CUDA_CHECK( hipEventSynchronize(stop) );
	MY_CUDA_CHECK( hipEventElapsedTime(&et, start, stop) );

out:

   TIMER_START(2);
   MY_CUDA_CHECK( hipMemcpy(hSRbuf, d_fsbuf, ncol*sizeof(*d_fsbuf), hipMemcpyDeviceToHost ));

   TIMER_STOP(2);

   return ncol;

}

/*
 * scan_frt_csc_cuda(frt, ncol, depth, hRFbuf);
 */
LOCINT scan_frt_csc_cuda_mono(int offset, int ncol, int depth) {

	LOCINT	i;
	int blocks, nrow=0;
	float	et=0;
	TIMER_DEF(1);


	LOCINT *d_ncbuf;

	TIMER_START(1);
	if (!ncol) {
		TIMER_STOP(1);
		goto out;
	}

#ifdef ONEPREFIX
	nrow = tlvl[depth+1];
	d_ncbuf = d_cbuf_start+offset;
#else

#ifdef THRUST

	static	thrust::device_ptr<LOCINT> d_val(d_cbuf);
#endif
	// calculate degree for each vertex in frt
	read_edge_count<<<(ncol+THREADS-1)/THREADS, THREADS>>>(d_deg, d_frt_start+offset, ncol, d_cbuf);

	MY_CUDA_CHECK( hipMemcpy(&i, d_cbuf+ncol-1, sizeof(*d_cbuf), hipMemcpyDeviceToHost) );
	nrow = i;

	// Prefix sum to count how many threads to launch
#ifdef THRUST
	thrust::exclusive_scan(d_val, d_val+ncol, d_val);
#else
	hipcub::DeviceScan::ExclusiveSum(d_cubtmp, cubtmp_sz, d_cbuf, d_cbuf, ncol);
#endif
	MY_CUDA_CHECK( hipMemcpy(&i, d_cbuf+ncol-1, sizeof(*d_cbuf), hipMemcpyDeviceToHost) );

	nrow += i;
	d_ncbuf = d_cbuf;
#endif

	if (!nrow) {
		TIMER_STOP(1);
		goto out;
	}
	TIMER_STOP(1);

	MY_CUDA_CHECK( hipEventRecord(start, 0) );

	blocks = (((nrow+ROWXTHD-1)/ROWXTHD)+THREADS-1)/THREADS;
	//dump_device_farray2(d_delta, row_pp, "d_delta");

	// Store result directly into d_delta
	scan_frt_mono<<<blocks, THREADS>>>(d_row, d_col, nrow, d_frt_start+offset, d_ncbuf, ncol,
			                           d_sig, d_delta, d_lvl, depth);

	MY_CUDA_CHECK( hipEventRecord(stop, 0) );
	MY_CHECK_ERROR("scan_frt");
	MY_CUDA_CHECK( hipEventSynchronize(stop) );
	MY_CUDA_CHECK( hipEventElapsedTime(&et, start, stop) );

out:

   return ncol;

}





/**
 */
LOCINT scan_col_csc_cuda_mono(int ncol, int level) {

	int blocks;
	LOCINT	i;
	float	et=0;
	LOCINT	nfrt=0, nrow=0;
#ifdef ONEPREFIX

#ifdef THRUST
	thrust::device_ptr<LOCINT> d_val(d_cbuf);
#endif
	int *d_out = NULL;

#else

#ifdef THRUST
    	static thrust::device_ptr<LOCINT> d_val(d_cbuf);
#endif

#endif

	TIMER_DEF(1);
	TIMER_DEF(2);

	TIMER_START(1);

	MY_CUDA_CHECK( hipMemset(d_snum, 0, sizeof(*d_snum)) );

	read_edge_count<<<(ncol+THREADS-1)/THREADS, THREADS>>>(d_deg, d_frt, ncol, d_cbuf);
	MY_CHECK_ERROR("read_edge_count");
	MY_CUDA_CHECK( hipDeviceSynchronize() );

	MY_CUDA_CHECK( hipMemcpy(&i, d_cbuf+ncol-1, sizeof(*d_cbuf), hipMemcpyDeviceToHost) );

	nrow = i;


	// Prefix sum to count how many threads to launch
#ifdef THRUST
	thrust::exclusive_scan(d_val, d_val+ncol, d_val);
#else
	hipcub::DeviceScan::ExclusiveSum(d_cubtmp, cubtmp_sz, d_cbuf, d_cbuf, ncol);
#endif
	MY_CUDA_CHECK( hipMemcpy(&i, d_cbuf+ncol-1, sizeof(*d_cbuf), hipMemcpyDeviceToHost) );
	nrow += i;

#ifdef ONEPREFIX
	tlvl[level] = nrow;
#endif

	if (!nrow) {
		TIMER_STOP(1);
		goto out;
	}
	TIMER_STOP(1);

	MY_CUDA_CHECK( hipEventRecord(start, 0) );

	blocks = (((nrow+ROWXTH-1)/ROWXTH)+THREADS-1)/THREADS;

	//dump_uarray2(&nrow,1,"scan_col nrow");
	//dump_device_array2(d_lvl, row_pp, "d_lvl");

	scan_col_mono2<<<blocks, THREADS>>>(d_row, d_col, nrow, d_frt, d_cbuf, ncol, d_msk, d_lvl, 
                                        d_sig, level, d_frt+ncol, d_snum);

	// Here we have d_sbuf updated with the new discovered vertices and d_tmp_sig with the local value of the accumulated sigma
	MY_CUDA_CHECK( hipEventRecord(stop, 0) );
	MY_CHECK_ERROR("scan_col");
	MY_CUDA_CHECK( hipEventSynchronize(stop) );
	MY_CUDA_CHECK( hipEventElapsedTime(&et, start, stop) );

	//dump_device_uarray2(d_sig, row_pp, "scan_col d_sig 2");
out:
	TIMER_START(2);

	// Prepare sbuf to send vertices to other processors (We need to send Sigma as well
	// copy d_snum back into CPU
	MY_CUDA_CHECK( hipMemcpy(&nfrt, d_snum, sizeof(nfrt), hipMemcpyDeviceToHost) );
	//dump_device_uarray2(d_frt+ncol, nfrt, "scan_col d_frt 3");

	d_frt = d_frt + ncol;
#ifdef ONEPREFIX
	d_cbuf = d_cbuf + ncol;
#endif

	TIMER_STOP(2);

	return nfrt;
}

/**
 * Note: np = number of all processors
 *       rowp = number of processors in a row
 */
LOCINT scan_col_csc_cuda(LOCINT *rbuf, LOCINT ld, int *rnum, int np, LOCINT *sbuf, int *snum,
		                 LOCINT *frt, LOCINT *frt_sig, int level) {

	int blocks;
	LOCINT	i,k;
	float	et=0;
	LOCINT	nfrt=0, ncol=0, nrow=0;
#ifdef THRUST
	static	thrust::device_ptr<LOCINT> d_val(d_cbuf);
#endif

	TIMER_DEF(1);
	TIMER_DEF(2);

	TIMER_START(1);

	MY_CUDA_CHECK( hipMemset(d_snum, 0, C*sizeof(*d_snum)) );

	k = 0;
	ncol = 0;

	LOCINT ld2 = ld*2;

	// Loop over the processors over the same Column (which are R processors)
	for(i = 0; i < np; i++) {
		if (rnum[i]) {
			// Note that while in CPU we use the same buffers for both Vertices and Sigma
			// in GPU we use two separate arrays

			// Copy in cuda the vertices received asynchronously using 2 streams (0,1)
			MY_CUDA_CHECK( hipMemcpyAsync(d_rbuf+ncol, rbuf+i*ld2, rnum[i]*sizeof(*rbuf), hipMemcpyHostToDevice, stream[k]) );
			// Copy in cuda the vertices sigma values received
			MY_CUDA_CHECK( hipMemcpyAsync(d_rbuf_sig+ncol, rbuf+i*ld2+rnum[i], rnum[i]*sizeof(*rbuf), hipMemcpyHostToDevice, stream[k]) );
			read_edge_count<<<(rnum[i]+THREADS-1)/THREADS, THREADS, 0, stream[k]>>>(d_deg, d_rbuf+ncol, rnum[i], d_cbuf+ncol);
			k ^= 1;
			ncol += rnum[i];
		}
	}
	MY_CHECK_ERROR("read_edge_count");
	MY_CUDA_CHECK( hipDeviceSynchronize() );
	// Here I have in d_rbuf all the vertices received and in d_deg the local degree for each of them
	// While in d_rbuf_sig we have all sigma values
	// ncol is the total number of vertices received
	if (ncol==0) {
		TIMER_STOP(1);
		goto out;
	}

	// dump_device_uarray2(d_rbuf, MAX(col_bl, row_pp) , "scan_col d_rbuf (1) ");
	//dump_device_uarray2(d_rbuf_sig, MAX(col_bl, row_pp) , "scan_col d_rbuf_sig (1) ");

	//dump_device_uarray2(d_cbuf, MAX(col_bl, C), "d_cbuf 1");

	MY_CUDA_CHECK( hipMemcpy(&i, d_cbuf+ncol-1, sizeof(*d_cbuf), hipMemcpyDeviceToHost) );

	nrow = i;

	// Prefix sum to count how many threads to launch
#ifdef THRUST
	thrust::exclusive_scan(d_val, d_val+ncol, d_val);
#else
	hipcub::DeviceScan::ExclusiveSum(d_cubtmp, cubtmp_sz, d_cbuf, d_cbuf, ncol);
#endif
	MY_CUDA_CHECK( hipMemcpy(&i, d_cbuf+ncol-1, sizeof(*d_cbuf), hipMemcpyDeviceToHost) );
	nrow += i;
	//dump_device_uarray2(d_cbuf, MAX(col_bl, C), "d_cbuf 2");
	if (!nrow) {
		TIMER_STOP(1);
		goto out;
	}
	TIMER_STOP(1);

	//dump_device_uarray2(d_sbuf, row_pp, "scan_col d_sbuf 1 ");
	//dump_device_uarray2(d_sbuf_sig, row_pp, "scan_col d_sbuf_sig 1 ");
	//dump_device_uarray2(d_tmp_sig, row_pp, "scan_col d_tmp_sig 1");

	MY_CUDA_CHECK( hipEventRecord(start, 0) );

	blocks = (((nrow+ROWXTH-1)/ROWXTH)+THREADS-1)/THREADS;

	//dump_device_uarray2(d_cbuf, MAX(col_bl, C), "d_cbuf");
	//dump_uarray2(&nrow,1,"scan_col nrow");
	//dump_device_array2(d_prd, row_pp, "d_prd");
	//dump_device_array2(d_lvl, row_pp, "d_lvl");
//sostituto d_prd con d_reach 9 arg
	scan_col<<<blocks, THREADS>>>(d_row, d_col, nrow, d_rbuf, d_rbuf_sig, d_cbuf, ncol, d_msk, d_lvl,
								  d_tmp_sig, level, d_sbuf, d_snum);

	// Here we have d_sbuf updated with the new discovered vertices and d_tmp_sig with the local value of the accumulated sigma
	MY_CUDA_CHECK( hipEventRecord(stop, 0) );
	MY_CHECK_ERROR("scan_col");
	MY_CUDA_CHECK( hipEventSynchronize(stop) );
	MY_CUDA_CHECK( hipEventElapsedTime(&et, start, stop) );

	//dump_device_uarray2(d_sig, row_pp, "scan_col d_sig 2");
	//dump_device_uarray2(d_sbuf, row_pp, "scan_col d_sbuf 2");
	//dump_device_uarray2(d_snum, C, "d_snum");
out:
	TIMER_START(2);

	// Prepare sbuf to send vertices to other processors (We need to send Sigma as well
	// copy d_snum back into CPU
	MY_CUDA_CHECK( hipMemcpy(snum, d_snum, C*sizeof(*snum), hipMemcpyDeviceToHost) );

	// Loop over the processors over the same Column (which are R processors)
	for(i = 0; i < C; i++) {
		//dump_device_array2(d_sbuf+i*ld, snum[i], "scan_col d_sbuf (3a) ");
		 if (snum[i]) {
			// Write in d_sbuf_sigma sigma values to be sent
			write_sigma<<<(snum[i]+THREADS-1)/THREADS, THREADS, 0, stream[k]>>>(d_sbuf+i*ld, d_sig, d_tmp_sig, snum[i], d_sbuf_sig+i*ld);

			// Copy into CPU 'sbuf' vertices to send in different position for each processor, but if they are for
		    // this processor copy into 'frt'
			// Sigma values are copied into sbuf as well while for this processor are copied into frt_sigma

			if (i!=mycol) {
				MY_CUDA_CHECK( hipMemcpyAsync(sbuf+i*ld2, d_sbuf+i*ld, snum[i]*sizeof(*d_sbuf), hipMemcpyDeviceToHost, stream[k]) );
				MY_CUDA_CHECK( hipMemcpyAsync(sbuf+i*ld2+snum[i], d_sbuf_sig+i*ld, snum[i]*sizeof(*d_sbuf_sig), hipMemcpyDeviceToHost, stream[k]) );
			}
			k ^= 1;
		}
	}
	MY_CHECK_ERROR("write_sigma");
	MY_CUDA_CHECK( hipDeviceSynchronize() );

	nfrt = snum[mycol];

	// ASSIGN_DFRT
	d_frt = d_sbuf + mycol*ld;
	d_frt_sig = d_sbuf_sig + mycol*ld;
	snum[mycol] = 0;

	TIMER_STOP(2);
/*
	prstat(ncol, "length of search array:", 1);
	prstat(((nrow+ROWXTH-1)/ROWXTH), "number of binsearch:", 1);
	prstat(TIMER_ELAPSED(1), "scan_col_csc_cuda 1 (us):", 1);
        prstat((uint64_t)(et*1000.0), "scan_col_csc_cuda K (us):", 1);
        prstat(TIMER_ELAPSED(2), "scan_col_csc_cuda 2 (us):", 1);
*/
	return nfrt;
}
		
LOCINT append_rows_cuda(LOCINT *rbuf, LOCINT ld,   int *rnum, int np,
			            LOCINT *frt, LOCINT *frt_sigma, LOCINT nfrt, int level) {
	float	et=0;
	LOCINT	nrow=0;
	LOCINT	p, q;

	LOCINT ld2 = ld*2;

	TIMER_DEF(1);
	
	TIMER_START(1);

	nrow = 0;
	for(int i = 0; i < np; i++) {
		if (rnum[i]) {
			MY_CUDA_CHECK( hipMemcpy(d_rbuf+nrow, rbuf+i*ld2, rnum[i]*sizeof(*rbuf), hipMemcpyHostToDevice) );
			MY_CUDA_CHECK( hipMemcpy(d_rbuf_sig+nrow, rbuf+i*ld2+rnum[i], rnum[i]*sizeof(*rbuf), hipMemcpyHostToDevice) );
			nrow += rnum[i];
		}
	}

	if (nrow > 0) {
		//	MY_CUDA_CHECK( hipMemcpy(d_rbuf, rbuf, nrow*sizeof(*rbuf), hipMemcpyHostToDevice) );
		// in-place prefix-sum of rnum (too small to bother thrust)
		p = rnum[0]; rnum[0] = 0;
		for(int i = 1; i < np; i++) {
			q = rnum[i];
			rnum[i] = p + rnum[i-i];
			p = q;
		}
		MY_CUDA_CHECK( hipMemcpy(d_cbuf, rnum, np*sizeof(*rnum), hipMemcpyHostToDevice) );
		MY_CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL(dnfrt), &nfrt, sizeof(dnfrt), 0, hipMemcpyHostToDevice) );
		TIMER_STOP(1);

		//dump_device_array2(d_rbuf_sig, nrow, "append d_rbuf_sig");
		//dump_device_array2(d_frt, nfrt, "append d_frt 1 ");
		//dump_device_array2(d_frt_sig, nfrt, "append d_frt_sig 1 ");
		//dump_device_array2(d_tmp_sig, row_pp, "append d_tmp_sig 1");

		MY_CUDA_CHECK( hipEventRecord(start, 0) );
		// Here update d_sbuf and d_sig, after that we need to update d_sbuf_sig

		// UPDATE DFRT sostituito d_pred con di reach... 7 arg
		append_row<<<(nrow+THREADS-1)/THREADS, THREADS>>>(d_rbuf, d_rbuf_sig, nrow, d_cbuf, np,
								  d_msk, d_reach, d_lvl, level, d_frt, d_tmp_sig, d_frt_sig, d_all);
		MY_CUDA_CHECK( hipEventRecord(stop, 0) );
		MY_CHECK_ERROR("append_row");
		MY_CUDA_CHECK( hipEventSynchronize(stop) );
		MY_CUDA_CHECK( hipEventElapsedTime(&et, start, stop) );
		//if (myid == 0) fprintf(stdout, "\tappend_row time = %f + %f\n", TIMER_ELAPSED(1)/1.0E+6, et/1.0E3);
	
		MY_CUDA_CHECK( hipMemcpyFromSymbol(&nfrt, HIP_SYMBOL(dnfrt), sizeof(nfrt), 0, hipMemcpyDeviceToHost) );

	}

	if (nfrt > 0) {

		MY_CUDA_CHECK( hipEventRecord(start, 0) );

		// READ DFRT
	    append_sigma<<<(nfrt+THREADS-1)/THREADS, THREADS>>>(d_frt, d_sig, d_frt_sig, d_tmp_sig, nfrt);

	    MY_CUDA_CHECK( hipEventRecord(stop, 0) );
	    MY_CHECK_ERROR("append_sigma");
	    MY_CUDA_CHECK( hipEventSynchronize(stop) );
	    MY_CUDA_CHECK( hipEventElapsedTime(&et, start, stop) );

	   // Add new vertices to the frontier
	   MY_CUDA_CHECK( hipMemcpy(frt, d_frt, nfrt*sizeof(*d_frt), hipMemcpyDeviceToHost) );
	   MY_CUDA_CHECK( hipMemcpy(frt_sigma, d_frt_sig, nfrt*sizeof(*d_frt_sig), hipMemcpyDeviceToHost) );
	}
	//dump_device_array2(d_frt, nfrt, "append d_frt 3");
	//dump_device_array2(d_frt_sig, nfrt, "append d_frt_sig 3");
	//dump_device_array2(d_tmp_sig, row_pp, "append d_sbuf_sig 3");
	//dump_device_array2(d_sig, row_pp, "append d_sig 3");
	
	return nfrt;
}

void set_get_overlap(LOCINT *sigma, int *lvl)
{

	//stream0 H2D
	//stream1 D2H
	// start for sigma
	// stop for lvl
	MY_CUDA_CHECK( hipMemcpyAsync(sigma+(mycol*row_bl), d_sig+(mycol*row_bl), row_bl*sizeof(*d_sig), hipMemcpyDeviceToHost, stream[0]) );
	hipEventRecord(start, stream[0]);
	MY_CUDA_CHECK( hipMemcpyAsync(lvl+(mycol*row_bl), d_lvl+(mycol*row_bl), row_bl*sizeof(*d_lvl), hipMemcpyDeviceToHost,stream[0]) );
	hipEventRecord(stop, stream[0]);

	hipEventSynchronize(start);
	//MPI_Allreduce(MPI_IN_PLACE, sigma, row_pp, LOCINT_MPI, MPI_SUM, Row_comm);
    MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, sigma, row_bl, LOCINT_MPI, Row_comm);
	MY_CUDA_CHECK( hipMemcpyAsync(d_sig, sigma, row_pp*sizeof(*sigma), hipMemcpyHostToDevice, stream[1]) ); 
	hipEventSynchronize(stop);
	//MPI_Allreduce(MPI_IN_PLACE, lvl, row_pp, MPI_INT, MPI_SUM, Row_comm);
	MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, lvl, row_bl, MPI_INT, Row_comm);
	MY_CUDA_CHECK( hipMemcpyAsync(d_lvl, lvl, row_pp*sizeof(*lvl), hipMemcpyHostToDevice, stream[1]) );
	
//        hipEventRecord(blahblah, 0);i	

}


void get_lvl(int *lvl) {
	MY_CUDA_CHECK( hipMemcpy(lvl+(mycol*row_bl), d_lvl+(mycol*row_bl), row_bl*sizeof(*d_lvl), hipMemcpyDeviceToHost) );
}

void set_lvl(int *lvl) {
	MY_CUDA_CHECK( hipMemcpy(d_lvl, lvl, row_pp*sizeof(*lvl), hipMemcpyHostToDevice) );
}

void get_all(LOCINT *all) {
	MY_CUDA_CHECK( hipMemcpy(all, d_all, sizeof(*d_all), hipMemcpyDeviceToHost) );
	// and set to zero for next bc
	MY_CUDA_CHECK(hipMemset(d_all, 0,sizeof(*d_all)));

}
	
void get_frt(LOCINT *frt) {
	MY_CUDA_CHECK( hipMemcpy(frt, d_frt_start, row_pp*sizeof(LOCINT), hipMemcpyDeviceToHost) );
}

void get_cbuf(LOCINT *cbuf) {
	MY_CUDA_CHECK( hipMemcpy(cbuf, d_cbuf_start, row_pp*sizeof(LOCINT), hipMemcpyDeviceToHost) );
}

void get_msk(LOCINT *msk) {
	MY_CUDA_CHECK( hipMemcpy(msk, d_msk, ((row_pp+BITS(d_msk)-1)/BITS(d_msk))*sizeof(*d_msk), hipMemcpyDeviceToHost) );
}

void get_deg(LOCINT *deg) {
	MY_CUDA_CHECK( hipMemcpy(deg, d_deg, col_bl*sizeof(*d_deg), hipMemcpyDeviceToHost) );
}

void get_sigma(LOCINT *sigma) {
	MY_CUDA_CHECK( hipMemcpy(sigma+(mycol*row_bl), d_sig+(mycol*row_bl), row_bl*sizeof(*d_sig), hipMemcpyDeviceToHost) );
//        MY_CUDA_CHECK( hipEventRecord( get_sigma_event, d2h_stream ) );
}

void get_bc(float *bc) {
	MY_CUDA_CHECK( hipMemcpy(bc, d_bc, row_pp*sizeof(*d_bc), hipMemcpyDeviceToHost) );
}

void get_delta(float *delta){
	MY_CUDA_CHECK( hipMemcpy(delta, d_delta, row_pp*sizeof(*d_delta), hipMemcpyDeviceToHost) );
}

void set_sigma(LOCINT *sigma) {
	MY_CUDA_CHECK( hipMemcpy(d_sig, sigma, row_pp*sizeof(*sigma), hipMemcpyHostToDevice) );
}

__global__ void set_delta(float *srbuf, float * delta, int nrow) {

	const uint32_t tid = blockDim.x*blockIdx.x + threadIdx.x;

	if (tid >= nrow) return;

	delta[tid] += srbuf[tid];
	srbuf[tid] = 0;

	return;
}

int set_delta_cuda(float *hSRbuf, int nrow) {

	float et=0;
	TIMER_DEF(1);

	TIMER_START(1);

	if (!nrow) {
		TIMER_STOP(1);
		return nrow;
	}

	//MY_CUDA_CHECK( hipMemset(d_frbuf, 0, row_pp*sizeof(*d_frbuf)) );
	MY_CUDA_CHECK( hipMemcpy(d_frbuf, hSRbuf , nrow*sizeof(*hSRbuf), hipMemcpyHostToDevice ));
	MY_CUDA_CHECK( hipEventRecord(start, 0) );
	set_delta<<<(nrow+THREADS-1)/THREADS, THREADS>>>(d_frbuf, d_delta, nrow);
	MY_CUDA_CHECK( hipEventRecord(stop, 0) );
	MY_CHECK_ERROR("set_delta");
	MY_CUDA_CHECK( hipEventSynchronize(stop) );
	MY_CUDA_CHECK( hipEventElapsedTime(&et, start, stop) );

	return nrow;
}

void init_bc_1degree_device(LOCINT *reach) {
      //MY_CUDA_CHECK( hipMemcpy(d_bc, bc_val,  row_pp*sizeof(*bc_val), hipMemcpyHostToDevice) );
      MY_CUDA_CHECK( hipMemcpy(d_reach, reach, row_pp*sizeof(*reach), hipMemcpyHostToDevice) );
      return;
}

__global__ void init_delta(LOCINT *reach, float * delta, int nrow) {

	const uint32_t tid = blockDim.x*blockIdx.x + threadIdx.x;

	if (tid >= nrow) return;

	delta[tid] = (float)reach[tid];

	return;
}

void setcuda(uint64_t ned, LOCINT *col, LOCINT *row, LOCINT reach_v0) {

	MY_CUDA_CHECK( hipMemset(d_lvl,-1, row_pp*sizeof(*d_lvl)) );
	MY_CUDA_CHECK( hipMemset(d_sig, 0, row_pp*sizeof(*d_sig)) );

	if (!mono) // Run Multi-GPU code
		MY_CUDA_CHECK( hipMemset(d_tmp_sig, 0, row_pp*sizeof(*d_tmp_sig)) );

	init_delta<<<(row_pp+THREADS-1)/THREADS, THREADS>>>(d_reach, d_delta, row_pp);

	MY_CUDA_CHECK( hipMemset(d_msk, 0, ((row_pp+BITS(d_msk)-1)/BITS(d_msk))*sizeof(*d_msk)) );

	MY_CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL(d_reach_v0), &reach_v0, sizeof(d_reach_v0), 0, hipMemcpyHostToDevice) );

#ifdef ONEPREFIX
	memset(tlvl, 0, sizeof(*tlvl)*MAX_LVL);
	d_cbuf = d_cbuf_start;
#endif

	d_frt = d_frt_start;

	return;
}



size_t initcuda(uint64_t ned, LOCINT *col, LOCINT *row) {

	int dev;

	dev = assignDeviceToProcess();
	MY_CUDA_CHECK( hipSetDevice(dev) );

	d_col = (LOCINT *)CudaMallocSet((col_bl+1)*sizeof(*d_col), 0);
	d_row = (LOCINT *)CudaMallocSet(ned*sizeof(*d_row), 0);
	MY_CUDA_CHECK( hipMemcpy(d_col, col, (col_bl+1)*sizeof(*col), hipMemcpyHostToDevice) );
	MY_CUDA_CHECK( hipMemcpy(d_row, row, ned*sizeof(*row), hipMemcpyHostToDevice) );

	d_deg = (LOCINT *)CudaMallocSet(col_bl*sizeof(*d_deg), 0); 
	set_degree<<<(col_bl+THREADS-1)/THREADS, THREADS>>>(d_col, d_deg, col_bl);

	if (!mono) { // Run Multi-GPU code
		d_rbuf =      (LOCINT *)CudaMallocSet(MAX(col_bl, row_pp)*sizeof(*d_rbuf), 0);
		d_rbuf_sig =  (LOCINT *)CudaMallocSet(MAX(col_bl, row_pp)*sizeof(*d_rbuf_sig), 0);
		d_sbuf_sig =  (LOCINT *)CudaMallocSet(MAX(row_bl,row_pp)*sizeof(*d_sbuf_sig), 0);
		d_tmp_sig =   (LOCINT *)CudaMallocSet(row_pp*sizeof(*d_tmp_sig), 0);
		d_fsbuf =     (float *)CudaMallocSet(MAX(row_pp, col_bl)*sizeof(*d_fsbuf), 0);
		d_frbuf =     (float *)CudaMallocSet(MAX(row_pp, col_bl)*sizeof(*d_frbuf), 0);
		d_cbuf_start = (LOCINT *)CudaMallocSet(MAX(col_bl, C)*sizeof(*d_cbuf_start), 0);
	} else {
#ifdef ONEPREFIX
		d_cbuf_start = (LOCINT *)CudaMallocSet(MAX(row_bl,row_pp)*sizeof(*d_cbuf_start), 0);

#else
		d_cbuf_start = (LOCINT *)CudaMallocSet(MAX(col_bl, C)*sizeof(*d_cbuf_start), 0);
#endif
	}
	
	d_sbuf = (LOCINT *)CudaMallocSet(MAX(row_bl,row_pp)*sizeof(*d_sbuf), 0);
	d_snum = (uint32_t *)CudaMallocSet((C+1)*sizeof(*d_snum), 0);

	d_msk = (LOCINT *)CudaMallocSet(((row_pp+BITS(d_msk)-1)/BITS(d_msk))*sizeof(*d_msk), 0);
	d_lvl = (int *)CudaMallocSet(row_pp*sizeof(*d_lvl), -1);
	d_all = (uint32_t *)CudaMallocSet(1*sizeof(*d_all), 0);

	d_frt = d_sbuf;
	d_frt_start = d_sbuf;
	d_cbuf = d_cbuf_start;

	d_sig =       (LOCINT *)CudaMallocSet(row_pp*sizeof(*d_sig), 0);
	
	d_delta =     (float *)CudaMallocSet(row_pp*sizeof(*d_delta), 0);
	d_bc    =     (float *)CudaMallocSet(row_pp*sizeof(*d_bc), 0);
        d_reach =     (LOCINT*)CudaMallocSet(row_pp*sizeof(*d_reach), 0);



#ifndef THRUST
	//CUB 
	size_t  bytes=0;
	hipcub::DeviceScan::ExclusiveSum(NULL, bytes, d_cbuf, d_cbuf, MAX(col_bl, C));	
//	hipcub::DeviceScan::ExclusiveSum(NULL, bytes, d_cbuf, d_cbuf, row_pp);	

	cubtmp_sz = bytes;

	d_cubtmp = (LOCINT *)CudaMallocSet(cubtmp_sz, 0);
#endif
	printf("ROWBL = %i - ROWPP = %i\n",row_bl,row_pp);

	MY_CUDA_CHECK( hipEventCreate(&start) );
    	MY_CUDA_CHECK( hipEventCreate(&stop) );

	MY_CUDA_CHECK( hipStreamCreate(stream+0) );
    	MY_CUDA_CHECK( hipStreamCreate(stream+1) );
	MY_CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL(dN), &N, sizeof(dN),  0, hipMemcpyHostToDevice) );
	MY_CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL(dC), &C, sizeof(dC),  0, hipMemcpyHostToDevice) );
	MY_CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL(dR), &R, sizeof(dR),  0, hipMemcpyHostToDevice) );
	MY_CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL(dmyrow), &myrow, sizeof(dmyrow),  0, hipMemcpyHostToDevice) );
	MY_CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL(dmycol), &mycol, sizeof(dmycol),  0, hipMemcpyHostToDevice) );
	MY_CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL(drow_bl), &row_bl, sizeof(drow_bl),  0, hipMemcpyHostToDevice) );
	MY_CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL(dcol_bl), &col_bl, sizeof(dcol_bl),  0, hipMemcpyHostToDevice) );
	MY_CUDA_CHECK( hipMemcpyToSymbol(HIP_SYMBOL(drow_pp), &row_pp, sizeof(drow_pp),  0, hipMemcpyHostToDevice) );

        MY_CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(read_edge_count), hipFuncCachePreferL1) );
        MY_CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(update_bc), hipFuncCachePreferL1) );
        MY_CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(deviceReduceKernel), hipFuncCachePreferL1) );

        if (!mono){

              MY_CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(append_sigma), hipFuncCachePreferL1) );
              MY_CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(scan_frt), hipFuncCachePreferL1) );
              MY_CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(scan_col), hipFuncCachePreferL1) );
              MY_CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(write_delta), hipFuncCachePreferL1) );
              MY_CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(append_row), hipFuncCachePreferL1) );
              MY_CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(write_sigma), hipFuncCachePreferL1) );
        }
        else{
        // set cache mono
            MY_CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(scan_frt_mono), hipFuncCachePreferL1) );
            MY_CUDA_CHECK( hipFuncSetCacheConfig(reinterpret_cast<const void*>(scan_col_mono), hipFuncCachePreferL1) );

        }

	return tot_dev_mem;
}
		

void fincuda() {

	MY_CUDA_CHECK( hipFree(d_col) );
	MY_CUDA_CHECK( hipFree(d_deg) ); //////////////////////
	MY_CUDA_CHECK( hipFree(d_row) );
	MY_CUDA_CHECK( hipFree(d_rbuf) );
	MY_CUDA_CHECK( hipFree(d_cbuf_start) );

	MY_CUDA_CHECK( hipFree(d_sbuf) );
	MY_CUDA_CHECK( hipFree(d_snum) );
	MY_CUDA_CHECK( hipFree(d_msk) );
	MY_CUDA_CHECK( hipFree(d_lvl) );

	MY_CUDA_CHECK( hipFree(d_tmp_sig) );
	MY_CUDA_CHECK( hipFree(d_sig) );
	MY_CUDA_CHECK( hipFree(d_rbuf_sig) );
	MY_CUDA_CHECK( hipFree(d_sbuf_sig) );

	MY_CUDA_CHECK( hipFree(d_frbuf) );
	MY_CUDA_CHECK( hipFree(d_fsbuf) );
	MY_CUDA_CHECK( hipFree(d_delta) );
	MY_CUDA_CHECK( hipFree(d_bc) );
#ifndef THRUST
	MY_CUDA_CHECK( hipFree(d_cubtmp) );
#endif
	MY_CUDA_CHECK( hipEventDestroy(start) );
        MY_CUDA_CHECK( hipEventDestroy(stop) );

	MY_CUDA_CHECK( hipStreamDestroy(stream[0]) );
	MY_CUDA_CHECK( hipStreamDestroy(stream[1]) );
        

        
	return;
}
